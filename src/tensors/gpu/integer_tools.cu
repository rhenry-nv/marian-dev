#include "hip/hip_runtime.h"
#include "tensors/gpu/integer_tools.h"
#include "tensors/gpu/cuda_helpers.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/cutlass.h"
#include "cutlass/epilogue/thread/linear_combination_relu.h"

namespace marian {

namespace gpu {
    
namespace integer {

    /**************************CUTLASS code begins here***********************/
    inline const char * cutlassGetErrorString(cutlass::Status& status) {
        switch (status) {
            case cutlass::Status::kSuccess:
                return "Operation was successful.";
            case cutlass::Status::kErrorMisalignedOperand:
                return "Operands fail alignment requirements.";
            case cutlass::Status::kErrorInvalidLayout:
                return "Layout fails alignment requirement.";
            case cutlass::Status::kErrorInvalidProblem:
                return "Specified problem size is not supported by operator.";
            case cutlass::Status::kErrorNotSupported:
                return "Operation is not supported on current device.";
            case cutlass::Status::kErrorWorkspaceNull:
                return "The given workspace is null when it is required to be non-null";
            case cutlass::Status::kErrorInternal:
                return "An error within CUTLASS occurred.";
            case cutlass::Status::kInvalid:
                return "Status is unspecified.";
        }
        return "Unknown CUTLASS status. Update this section of the code.";
    }
    
    #define CUTLASS_CHECK(expr) do {                                                                        \
        cutlass::Status rc = (expr);                                                                        \
        ABORT_IF(rc != cutlass::Status::kSuccess,                                                           \
                    "Cutlass Error: {} - {}:{}: {}", cutlassGetErrorString(rc), __FILE__, __LINE__, #expr); \
        } while(0)
    
    /*Cutlass matrices*/
    using ElementOutput = float;
    using ElementAccumulator = int32_t;
    using ElementCompute = float;
    /*TensorOp matrices*/

#ifdef CUTLASS_SM75
    // Compute arch
    using SmArch = cutlass::arch::Sm75;
    // This code section describes the tile size a thread block will compute
    using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<128, 256, 64>;  // <- threadblock tile M = 128, N = 256, K = 64
    // This code section describes tile size a warp will compute
    using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 64>;  // <- warp tile M = 64, N = 64, K = 64 
    // This code section describes the size of MMA op
    using ShapeMMAOp = cutlass::gemm::GemmShape<8, 8, 16>;  // <- MMA Op tile M = 8, N = 8, K = 16
    // This code section describes how threadblocks are scheduled on GPU
    using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??
    // Number of pipelines you want to use
    constexpr int NumStages = 2;
#else
    // Compute arch
    using SmArch = cutlass::arch::Sm80;
    // This code section describes the tile size a thread block will compute
    using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<128, 128, 64>;  // <- threadblock tile M = 128, N = 128, K = 16
    // This code section describes tile size a warp will compute
    using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 64>;  // <- warp tile M = 64, N = 64, K = 16
    // This code section describes the size of MMA op
    using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 32>;  // <- MMA Op tile M = 16, N = 8, K = 8
    // This code section describes how threadblocks are scheduled on GPU
    using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??
    // Number of pipelines you want to use
    constexpr int NumStages = 3;
#endif

#if 0 // These settings are for A100 which have bigger shared memory. Otherwise they crash with the 3090tis that we have
    // Compute arch
    using SmArch = cutlass::arch::Sm80;
    // This code section describes the tile size a thread block will compute
    using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<256, 128, 128>;  // <- threadblock tile M = 128, N = 128, K = 16
    // This code section describes tile size a warp will compute
    using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 128>;  // <- warp tile M = 64, N = 64, K = 16
    // This code section describes the size of MMA op
    using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 32>;  // <- MMA Op tile M = 16, N = 8, K = 8
    // This code section describes how threadblocks are scheduled on GPU
    using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??
    // Number of pipelines you want to use
    constexpr int NumStages = 4;
#endif


    // This code section describes the epilogue part of the kernel
    using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
                                                  ElementOutput, // <- data type of output matrix
                                                  128 / cutlass::sizeof_bits<ElementOutput>::value,  // <- the number of elements per vectorized
                                                       // memory access. For a byte, it's 16
                                                       // elements. This becomes the vector width of
                                                       // math instructions in the epilogue too
                                                  ElementAccumulator, // <- data type of accumulator
                                                  ElementCompute>;  // <- data type for alpha/beta in linear combination function

    using EpilogueOpRelu = cutlass::epilogue::thread::LinearCombinationRelu<
                                                      ElementOutput, // <- data type of output matrix
                                                      128 / cutlass::sizeof_bits<ElementOutput>::value,  // <- the number of elements per vectorized
                                                                // memory access. For a byte, it's 16
                                                                // elements. This becomes the vector width of
                                                                // math instructions in the epilogue too
                                                      ElementAccumulator, // <- data type of accumulator
                                                      ElementCompute>;  // <- data type for alpha/beta in linear combination function

    using CutlassGemmTensorOp = cutlass::gemm::device::Gemm<int8_t,                            // ElementA
                                                            cutlass::layout::RowMajor,         // LayoutA
                                                            int8_t,                            // ElementB
                                                            cutlass::layout::ColumnMajor,      // LayoutB
                                                            float,                             // ElementOutput
                                                            cutlass::layout::ColumnMajor,      // LayoutOutput
                                                            int32_t,                           // ElementAccumulator
                                                            cutlass::arch::OpClassTensorOp,    // tag indicating Tensor Cores
                                                            SmArch,                            // tag indicating target GPU compute architecture //@TODO this should change, probably
                                                            ShapeMMAThreadBlock,
                                                            ShapeMMAWarp,
                                                            ShapeMMAOp,
                                                            EpilogueOp,
                                                            SwizzleThreadBlock,
                                                            NumStages>;
    using CutlassGemmTensorOpRelu = cutlass::gemm::device::Gemm<int8_t,                           // ElementA
                                                               cutlass::layout::RowMajor,         // LayoutA
                                                               int8_t,                            // ElementB
                                                               cutlass::layout::ColumnMajor,      // LayoutB
                                                               float,                             // ElementOutput
                                                               cutlass::layout::ColumnMajor,      // LayoutOutput
                                                               int32_t,                           // ElementAccumulator
                                                               cutlass::arch::OpClassTensorOp,    // tag indicating Tensor Cores
                                                               SmArch,                            // tag indicating target GPU compute architecture //@TODO this should change, probably
                                                               ShapeMMAThreadBlock,
                                                               ShapeMMAWarp,
                                                               ShapeMMAOp,
                                                               EpilogueOpRelu,
                                                               SwizzleThreadBlock,
                                                               NumStages>;
    /*Non TensorOp matrices*/
    using InstructionShape = cutlass::gemm::GemmShape<1, 1, 4>;
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 64, 16>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 16>;
    using Epilogue = cutlass::epilogue::thread::LinearCombination<ElementOutput,
                                                                  1, /*@TODO should be something different? like 32/64/128?*/
                                                                  ElementAccumulator,
                                                                  ElementCompute>;

    using EpilogueRelu = cutlass::epilogue::thread::LinearCombinationRelu<ElementOutput,
                                                                  1, /*@TODO should be something different? like 32/64/128?*/
                                                                  ElementAccumulator,
                                                                  ElementCompute>;

    using ColumnMajor = cutlass::layout::ColumnMajor;
    using ColumnMajorT = cutlass::layout::RowMajor; //Transposing in cutlass is done by changing the input from RowMajor to ColumnMajor. Care of the output
    //using RowMajor = cutlass::layout::RowMajor;
    using CutlassGemmTT = cutlass::gemm::device::Gemm<int8_t,       // Data-type of A matrix
                                                    ColumnMajorT,   // Layout of A matrix
                                                    int8_t,         // Data-type of B matrix
                                                    ColumnMajorT,   // Layout of B matrix
                                                    float,          // Data-type of C matrix
                                                    ColumnMajor,    // Layout of C matrix
                                                    int32_t,        // Accumulator
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm75,
                                                    ThreadBlockShape,
                                                    WarpShape,
                                                    InstructionShape,
                                                    Epilogue
                                                    >;
    using CutlassGemmNT = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                    ColumnMajor,  // Layout of A matrix
                                                    int8_t,        // Data-type of B matrix
                                                    ColumnMajorT,  // Layout of B matrix
                                                    float,        // Data-type of C matrix
                                                    ColumnMajor, // Layout of C matrix
                                                    int32_t,        // Accumulator
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm75,
                                                    ThreadBlockShape,
                                                    WarpShape,
                                                    InstructionShape,
                                                    Epilogue
                                                    >;

    using CutlassGemmTN = cutlass::gemm::device::Gemm<int8_t,       // Data-type of A matrix
                                                    ColumnMajorT,   // Layout of A matrix
                                                    int8_t,         // Data-type of B matrix
                                                    ColumnMajor,    // Layout of B matrix
                                                    float,          // Data-type of C matrix
                                                    ColumnMajor,    // Layout of C matrix
                                                    int32_t,        // Accumulator
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm75,
                                                    ThreadBlockShape,
                                                    WarpShape,
                                                    InstructionShape,
                                                    Epilogue
                                                    >;

    using CutlassGemmNN = cutlass::gemm::device::Gemm<int8_t,      // Data-type of A matrix
                                                    ColumnMajor,   // Layout of A matrix
                                                    int8_t,        // Data-type of B matrix
                                                    ColumnMajor,   // Layout of B matrix
                                                    float,         // Data-type of C matrix
                                                    ColumnMajor,   // Layout of C matrix
                                                    int32_t,       // Accumulator
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm75,
                                                    ThreadBlockShape,
                                                    WarpShape,
                                                    InstructionShape,
                                                    Epilogue
                                                    >;

    using CutlassGemmTTRelu = cutlass::gemm::device::Gemm<int8_t,       // Data-type of A matrix
                                                    ColumnMajorT,   // Layout of A matrix
                                                    int8_t,         // Data-type of B matrix
                                                    ColumnMajorT,   // Layout of B matrix
                                                    float,          // Data-type of C matrix
                                                    ColumnMajor,    // Layout of C matrix
                                                    int32_t,        // Accumulator
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm75,
                                                    ThreadBlockShape,
                                                    WarpShape,
                                                    InstructionShape,
                                                    EpilogueRelu
                                                    >;
    using CutlassGemmNTRelu = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                    ColumnMajor,  // Layout of A matrix
                                                    int8_t,        // Data-type of B matrix
                                                    ColumnMajorT,  // Layout of B matrix
                                                    float,        // Data-type of C matrix
                                                    ColumnMajor, // Layout of C matrix
                                                    int32_t,        // Accumulator
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm75,
                                                    ThreadBlockShape,
                                                    WarpShape,
                                                    InstructionShape,
                                                    EpilogueRelu
                                                    >;

    using CutlassGemmTNRelu = cutlass::gemm::device::Gemm<int8_t,       // Data-type of A matrix
                                                    ColumnMajorT,   // Layout of A matrix
                                                    int8_t,         // Data-type of B matrix
                                                    ColumnMajor,    // Layout of B matrix
                                                    float,          // Data-type of C matrix
                                                    ColumnMajor,    // Layout of C matrix
                                                    int32_t,        // Accumulator
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm75,
                                                    ThreadBlockShape,
                                                    WarpShape,
                                                    InstructionShape,
                                                    EpilogueRelu
                                                    >;

    using CutlassGemmNNRelu = cutlass::gemm::device::Gemm<int8_t,      // Data-type of A matrix
                                                    ColumnMajor,   // Layout of A matrix
                                                    int8_t,        // Data-type of B matrix
                                                    ColumnMajor,   // Layout of B matrix
                                                    float,         // Data-type of C matrix
                                                    ColumnMajor,   // Layout of C matrix
                                                    int32_t,       // Accumulator
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm75,
                                                    ThreadBlockShape,
                                                    WarpShape,
                                                    InstructionShape,
                                                    EpilogueRelu
                                                    >;

    /*Non-Epilogue functions, as they are faster (for now)*/
    using CutlassGemmTensorOpunfused = cutlass::gemm::device::Gemm<int8_t,                         // ElementA
                                                                cutlass::layout::RowMajor,         // LayoutA
                                                                int8_t,                            // ElementB
                                                                cutlass::layout::ColumnMajor,      // LayoutB
                                                                int32_t,                           // ElementOutput
                                                                cutlass::layout::ColumnMajor,      // LayoutOutput
                                                                int32_t,                           // ElementAccumulator
                                                                cutlass::arch::OpClassTensorOp,    // tag indicating Tensor Cores
                                                                cutlass::arch::Sm75>;

    using CutlassGemmTTunfused = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                            ColumnMajorT,   // Layout of A matrix
                                                            int8_t,         // Data-type of B matrix
                                                            ColumnMajorT,   // Layout of B matrix
                                                            int32_t,        // Data-type of C matrix
                                                            ColumnMajor,    // Layout of C matrix
                                                            int32_t>;       // Accumulator

    using CutlassGemmNTunfused = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                            ColumnMajor,    // Layout of A matrix
                                                            int8_t,         // Data-type of B matrix
                                                            ColumnMajorT,   // Layout of B matrix
                                                            int32_t,        // Data-type of C matrix
                                                            ColumnMajor,    // Layout of C matrix
                                                            int32_t>;       // Accumulator

    using CutlassGemmTNunfused = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                            ColumnMajorT,   // Layout of A matrix
                                                            int8_t,         // Data-type of B matrix
                                                            ColumnMajor,    // Layout of B matrix
                                                            int32_t,        // Data-type of C matrix
                                                            ColumnMajor,    // Layout of C matrix
                                                            int32_t>;       // Accumulator

    using CutlassGemmNNunfused = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                            ColumnMajor,    // Layout of A matrix
                                                            int8_t,         // Data-type of B matrix
                                                            ColumnMajor,    // Layout of B matrix
                                                            int32_t,        // Data-type of C matrix
                                                            ColumnMajor,    // Layout of C matrix
                                                            int32_t>;       // Accumulator

    cutlass::Status cutlass_igemm_nn(bool transA, bool transB,
                        int M,
                        int N,
                        int K,
                        float * alpha,
                        int8_t const *A,
                        int lda,
                        int8_t const *B,
                        int ldb,
                        float * beta,
                        float *C,
                        int ldc,
                        bool tensorCore, /*We want this to be true for best performance*/
                        bool fused,     /* fused unquantisation (and bias addition (and activation function) if those are present). Should be true for best performance */
                        float * bias,
                        bool doRelu) {
        //printf("Success M:%d N:%d K%d, Relu:%d, bias:%d\n", M, N, K, (int)doRelu, (int)(bias!=nullptr));
        float * Csrc;
        int ldcSRC;
        if (bias) { /* This is only available for the fused option. Beta needs to be 1? */
            Csrc = bias;
            ldcSRC = 0; /*Having a stride of 0 enables bias broadcast*/
        } else {
            Csrc = C;
            ldcSRC = ldc;
        }
        if (fused) {
            if (doRelu) {
                if (tensorCore) {
                    CutlassGemmTensorOpRelu gemm_operator;
                    CutlassGemmTensorOpRelu::Arguments args({M, N, K},  // Gemm Problem dimensions
                        {A, lda},       // Tensor-ref for source matrix A
                        {B, ldb},       // Tensor-ref for source matrix B
                        {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                        {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                        {alpha, beta}); // Scalars used in the Epilogue
                    return gemm_operator(args);
                } else {
                    if (!transA && !transB) {
                        CutlassGemmNNRelu gemm_operator;
                        CutlassGemmNNRelu::Arguments args({M, N, K},  // Gemm Problem dimensions
                            {A, lda},       // Tensor-ref for source matrix A
                            {B, ldb},       // Tensor-ref for source matrix B
                            {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                            {alpha, beta}); // Scalars used in the Epilogue
                        return gemm_operator(args);
                    } else if (transA && !transB) {
                        CutlassGemmTNRelu gemm_operator;
                        CutlassGemmTNRelu::Arguments args({M, N, K},  // Gemm Problem dimensions
                            {A, lda},       // Tensor-ref for source matrix A
                            {B, ldb},       // Tensor-ref for source matrix B
                            {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                            {alpha, beta}); // Scalars used in the Epilogue
                        return gemm_operator(args);
                    } else if (!transA && transB) {
                        CutlassGemmNTRelu gemm_operator;
                        CutlassGemmNTRelu::Arguments args({M, N, K},  // Gemm Problem dimensions
                            {A, lda},       // Tensor-ref for source matrix A
                            {B, ldb},       // Tensor-ref for source matrix B
                            {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                            {alpha, beta}); // Scalars used in the Epilogue
                        return gemm_operator(args);
                    } else { // Final case (transA && transB)
                        CutlassGemmTTRelu gemm_operator;
                        CutlassGemmTTRelu::Arguments args({M, N, K},  // Gemm Problem dimensions
                            {A, lda},       // Tensor-ref for source matrix A
                            {B, ldb},       // Tensor-ref for source matrix B
                            {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                            {alpha, beta}); // Scalars used in the Epilogue
                        return gemm_operator(args);
                    }
                }
            } else {
                if (tensorCore) {
                    CutlassGemmTensorOp gemm_operator;
                    CutlassGemmTensorOp::Arguments args({M, N, K},  // Gemm Problem dimensions
                        {A, lda},       // Tensor-ref for source matrix A
                        {B, ldb},       // Tensor-ref for source matrix B
                        {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                        {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                        {alpha, beta}); // Scalars used in the Epilogue
                    return gemm_operator(args);
                } else {
                    if (!transA && !transB) {
                        CutlassGemmNN gemm_operator;
                        CutlassGemmNN::Arguments args({M, N, K},  // Gemm Problem dimensions
                            {A, lda},       // Tensor-ref for source matrix A
                            {B, ldb},       // Tensor-ref for source matrix B
                            {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                            {alpha, beta}); // Scalars used in the Epilogue
                        return gemm_operator(args);
                    } else if (transA && !transB) {
                        CutlassGemmTN gemm_operator;
                        CutlassGemmTN::Arguments args({M, N, K},  // Gemm Problem dimensions
                            {A, lda},       // Tensor-ref for source matrix A
                            {B, ldb},       // Tensor-ref for source matrix B
                            {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                            {alpha, beta}); // Scalars used in the Epilogue
                        return gemm_operator(args);
                    } else if (!transA && transB) {
                        CutlassGemmNT gemm_operator;
                        CutlassGemmNT::Arguments args({M, N, K},  // Gemm Problem dimensions
                            {A, lda},       // Tensor-ref for source matrix A
                            {B, ldb},       // Tensor-ref for source matrix B
                            {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                            {alpha, beta}); // Scalars used in the Epilogue
                        return gemm_operator(args);
                    } else { // Final case (transA && transB)
                        CutlassGemmTT gemm_operator;
                        CutlassGemmTT::Arguments args({M, N, K},  // Gemm Problem dimensions
                            {A, lda},       // Tensor-ref for source matrix A
                            {B, ldb},       // Tensor-ref for source matrix B
                            {Csrc, ldcSRC}, // Tensor-ref for source matrix C
                            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                            {alpha, beta}); // Scalars used in the Epilogue
                        return gemm_operator(args);
                    }
                }
            }
        } else {
            static const int32_t constexpr alpha_int = 1;
            static const int32_t constexpr beta_int = 0;
            if (tensorCore) {
                CutlassGemmTensorOpunfused gemm_operator;
                CutlassGemmTensorOpunfused::Arguments args({M, N, K},  // Gemm Problem dimensions
                    {A, lda},    // Tensor-ref for source matrix A
                    {B, ldb},    // Tensor-ref for source matrix B
                    {(int32_t *)C, ldc},    // Tensor-ref for source matrix C
                    {(int32_t *)C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                    {alpha_int, beta_int}); // Scalars used in the Epilogue
                return gemm_operator(args);
            } else {
                if (!transA && !transB) {
                    CutlassGemmNNunfused gemm_operator;
                    CutlassGemmNNunfused::Arguments args({M, N, K},  // Gemm Problem dimensions
                        {A, lda},    // Tensor-ref for source matrix A
                        {B, ldb},    // Tensor-ref for source matrix B
                        {(int32_t *)C, ldc},    // Tensor-ref for source matrix C
                        {(int32_t *)C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                        {alpha_int, beta_int}); // Scalars used in the Epilogue
                    return gemm_operator(args);
                } else if (transA && !transB) {
                    CutlassGemmTNunfused gemm_operator;
                    CutlassGemmTNunfused::Arguments args({M, N, K},  // Gemm Problem dimensions
                        {A, lda},    // Tensor-ref for source matrix A
                        {B, ldb},    // Tensor-ref for source matrix B
                        {(int32_t *)C, ldc},    // Tensor-ref for source matrix C
                        {(int32_t *)C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                        {alpha_int, beta_int}); // Scalars used in the Epilogue
                    return gemm_operator(args);
                } else if (!transA && transB) {
                    CutlassGemmNTunfused gemm_operator;
                    CutlassGemmNTunfused::Arguments args({M, N, K},  // Gemm Problem dimensions
                        {A, lda},    // Tensor-ref for source matrix A
                        {B, ldb},    // Tensor-ref for source matrix B
                        {(int32_t *)C, ldc},    // Tensor-ref for source matrix C
                        {(int32_t *)C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                        {alpha_int, beta_int}); // Scalars used in the Epilogue
                    return gemm_operator(args);
                } else { // Final case (transA && transB)
                    CutlassGemmTTunfused gemm_operator;
                    CutlassGemmTTunfused::Arguments args({M, N, K},  // Gemm Problem dimensions
                        {A, lda},    // Tensor-ref for source matrix A
                        {B, ldb},    // Tensor-ref for source matrix B
                        {(int32_t *)C, ldc},    // Tensor-ref for source matrix C
                        {(int32_t *)C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                        {alpha_int, beta_int}); // Scalars used in the Epilogue
                    return gemm_operator(args);
                }
            }
        }
    }
    void cutlass_igemm_dispatcher(bool transA, bool transB,
        int M,
        int N,
        int K,
        float * alpha,
        int8_t const *A,
        int lda,
        int8_t const *B,
        int ldb,
        float * beta,
        int32_t *C,
        int ldc,
        bool tensorCore,
        bool fused,
        float * bias,
        bool doRelu) {
            CUTLASS_CHECK(cutlass_igemm_nn(transA, transB,
                M,
                N,
                K,
                alpha,
                A,
                lda,
                B,
                ldb,
                beta,
                (float *)C,
                ldc,
                tensorCore,
                fused,
                bias,
                doRelu));
        CUDA_CHECK(hipGetLastError()); // Sometimes CUTLASS errors manifest as CUDA errors.
    }
    /**************************CUTLASS code ends here***********************/
    __global__ void getMaxAbsKernel(const float * input_gpu, int idxMax, int idxMin, float * output) {
        float absMax = abs(input_gpu[idxMax]);
        float absMin = abs(input_gpu[idxMin]);
        if (absMax > absMin) {
            output[0] = absMax;
        } else {
            output[0] = absMin;
        }
    }

    float getMaxAbs(hipblasHandle_t& handle, const float * input_gpu, size_t items) {
        // Allocate memory on the GPU
        float * output_gpu;
        CUDA_CHECK(hipMallocManaged(&output_gpu, sizeof(float)));

        //Get Max Absolute:
        int resMaxIdx;
        CUBLAS_CHECK(hipblasIsamax(handle, items, input_gpu, 1, &resMaxIdx));
        int resMinIdx;
        CUBLAS_CHECK(hipblasIsamin(handle, items, input_gpu, 1, &resMinIdx));

        getMaxAbsKernel<<<1,1>>>(input_gpu, resMaxIdx - 1, resMinIdx - 1, output_gpu); //FUCK YOU FORTRAN INDEXING
        CUDA_CHECK(hipDeviceSynchronize()); // We need to synchronise in order to use the managed memory

        float ret = *output_gpu;
        hipFree(output_gpu);
        return ret;
    }

    __global__ void findMaxMinAndQuantMult(const float * input_gpu, int idxMax, int idxMin, float * output) {
        float absMax = abs(input_gpu[idxMax]);
        float absMin = abs(input_gpu[idxMin]);
        if (absMax > absMin) {
            output[0] = 127.0f/absMax;
        } else {
            output[0] = 127.0f/absMin;
        }
    }
    
    //@TODO rewrite with a nice singlePass GPU version that uses shared memory
    void maxAbsQuantMult(hipblasHandle_t& handle, const float * input_gpu, size_t items, float * output_gpu) {
        //Get Max Absolute:
        int resMaxIdx;
        CUBLAS_CHECK(hipblasIsamax(handle, items, input_gpu, 1, &resMaxIdx));
        int resMinIdx;
        CUBLAS_CHECK(hipblasIsamin(handle, items, input_gpu, 1, &resMinIdx));

        findMaxMinAndQuantMult<<<1,1>>>(input_gpu, resMaxIdx - 1, resMinIdx - 1, output_gpu); //FUCK YOU FORTRAN INDEXING
    }

    __global__ void quantize(const float * input, int8_t * output, size_t items, const float * quantMultAddr) {
        const float quantMult = *quantMultAddr; //@TODO ask nvidia if this is the most efficient way to do this here
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        int i = threadIdx.x;
        __shared__ float share[256]; // Not sure if shared memory is necessary here to take advnatage of globale memory burst
        if (x < items) {
            share[i] = input[x];
            output[x] = (int8_t)max(-128, min(127, (int)rintf(share[i]*quantMult)));
        }
    }

    void quantize(const float * input, int8_t * output, size_t rows, size_t cols, const float * quantMultAddr) {
        // Make sure we're not running out of threads here.
        int threads = 256;
        int blocks = (int)ceil(rows*cols/256);

        quantize<<<blocks, threads>>>(input, output, rows*cols, quantMultAddr);
        CUDA_CHECK(hipGetLastError()); // Get errors from kernel launches
    }

    __global__ void quantizeToRowMajor(const float * input, int8_t * output, size_t rows, size_t cols, const float * quantMultAddr) {
        const float quantMult = *quantMultAddr; // @TODO ask nvidia if this is the most efficient way to do this here
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        //input is col major, output is row major
        if (row*col < rows*cols) {
            output[cols*row + col] = (int8_t)llrintf((input[rows*col + row]*quantMult));
        }
    }

    void quantizeToRowMajorWrapper(const float * input, int8_t * output, size_t rows, size_t cols, const float * quantMultAddr) {
        // Make sure we're not running out of threads here.

        dim3 dimBlock(32, 32);
        dim3 dimGrid(std::max(cols / dimBlock.x, 1ul), std::max(rows / dimBlock.y, 1ul));
        //dim3 dimGrid(std::max(cols / dimBlock.x, 2ul), std::max(rows / dimBlock.y, 2ul));

        quantizeToRowMajor<<<dimGrid, dimBlock>>>(input, output, rows, cols, quantMultAddr);
        CUDA_CHECK(hipGetLastError()); // Get errors from kernel launches
    }

    __global__ void getDequantMult(float * output, float * quantMultAaddr, float * quantMultBaddr) {
        const float aQuantMult = *quantMultAaddr;
        const float bQuantMult = *quantMultBaddr;
        *output = 1.0f/(aQuantMult*bQuantMult);
    }

    void getDequantMultWrapper(float * output, float * quantMultAaddr, float * quantMultBaddr) {
        getDequantMult<<<1,1>>>(output, quantMultAaddr, quantMultBaddr);
    }

    __global__ void dequantize(const int32_t * input, float * output, size_t items, const float * quantMultAaddr, const float * quantMultBaddr) {
        const float aQuantMult = *quantMultAaddr;
        const float bQuantMult = *quantMultBaddr;
        const float dequantMult = 1.0f/(aQuantMult*bQuantMult); //@TODO ask nvidia if this is the most efficient way to do this here
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        int i = threadIdx.x;
        __shared__ int32_t share[256]; // Not sure if shared memory is necessary here to take advnatage of globale memory burst
        if (x < items) {
            share[i] = input[x];
            output[x] = ((float)share[i])*dequantMult;
        }
    }

    void dequantize(const int32_t * input, float * output, size_t rows, size_t cols, const float * quantMultAaddr, const float * quantMultBaddr) {
        // Make sure we're not running out of threads here.
        int threads = 256;
        int blocks = (int)ceil(rows*cols/256);

        dequantize<<<blocks, threads>>>(input, output, rows*cols, quantMultAaddr, quantMultBaddr);
        CUDA_CHECK(hipGetLastError()); // Get errors from kernel launches
    }

    __global__ void dequantize(const int32_t * input, float * output, size_t items, const float * dequantMultAddr) {
        const float dequantMult = *dequantMultAddr; //@TODO ask nvidia if this is the most efficient way to do this here
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        int i = threadIdx.x;
        __shared__ int32_t share[256]; // Not sure if shared memory is necessary here to take advnatage of global memory burst
        if (x < items) {
            share[i] = input[x];
            output[x] = ((float)share[i])*dequantMult;
        }
    }

    void dequantize(const int32_t * input, float * output, size_t rows, size_t cols, const float * dequantMultAddr) {
        // Make sure we're not running out of threads here.
        int threads = 256;
        int blocks = (int)ceil(rows*cols/256);

        dequantize<<<blocks, threads>>>(input, output, rows*cols, dequantMultAddr);
        CUDA_CHECK(hipGetLastError()); // Get errors from kernel launches
    }

    __global__ void meanStdkern(float * input, size_t elems, float * mean, float * stddev, float * absMean, float * absStddev, float * normal_sum, float * squares_sum, float * abs_normal_sum) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;

        float * global_sums[3];

        global_sums[0] = normal_sum;
        global_sums[1] = squares_sum;
        global_sums[2] = abs_normal_sum;

        // Initiate shared memory
        __shared__ float shared_sums[3];
        float * normal_sum_share = &shared_sums[0];
        float * squares_sum_share = &shared_sums[1];
        float * abs_normal_sum_share = &shared_sums[2];
        if (threadIdx.x < 3) {
            shared_sums[threadIdx.x] = 0.0f;
        }
        __syncthreads();

        // Compute sums
        if (i < elems) {
            float current = input[i];
            atomicAdd(normal_sum_share, current);
            if (!isfinite(*normal_sum_share)) {
                //printf("nan\\inf detected at line 517, sum %f current %f\n", *normal_sum_share, current);
            }
            atomicAdd(abs_normal_sum_share, fabs(current));
            if (!isfinite(*abs_normal_sum_share)) {
                //printf("nan\\inf detected at line 521, sum %f current %f\n", *abs_normal_sum_share, current);
            }
            atomicAdd(squares_sum_share, current*current);
            if (!isfinite(*squares_sum_share)) {
                //printf("nan\\inf detected at line 525, sum %f current %f\n", *squares_sum_share, current);
            }
        }
        __syncthreads();
        // The first three threads in each block should write to the global_memory
        if (threadIdx.x < 3) {
            atomicAdd(global_sums[threadIdx.x], shared_sums[threadIdx.x]);
            if (!isfinite(*global_sums[threadIdx.x])) {
                //printf("nan\\inf detected at line 533, i is: %d \n", (int)threadIdx.x);
            }
        }
        __syncthreads();

        // Write the results to global memory
        if (i == 0) {
            *mean = (*normal_sum)/elems;
            *stddev = sqrtf(((*squares_sum)/elems) - ((*mean)*(*mean)));
            if (!isfinite(((*mean)*(*mean)))) {
                //printf("nan\\inf detected at line 543\n");
            }
        } else if (i == 1) {
            *absMean = (*abs_normal_sum)/elems;
            *absStddev = sqrtf(((*squares_sum)/elems) - ((*absMean)*(*absMean)));
            if (!isfinite(((*absMean)*(*absMean)))) {
                //printf("nan\\inf detected at line 549\n");
            }
        }
    }

    MeanStd getMeanStd(float * input, size_t elems) {
        MeanStd ret;
        // Allocate GPU memory. Use CudaMallocManaged to avoid copy to CPU memory after
        float * mean;
        float * stddev;
        float * absMean;
        float * absStddev;
        float * normal_sum;
        float * squares_sum;
        float * abs_normal_sum;
        CUDA_CHECK(hipMallocManaged(&mean, sizeof(float)));
        CUDA_CHECK(hipMallocManaged(&stddev, sizeof(float)));
        CUDA_CHECK(hipMallocManaged(&absMean, sizeof(float)));
        CUDA_CHECK(hipMallocManaged(&absStddev, sizeof(float)));
        CUDA_CHECK(hipMallocManaged(&normal_sum, sizeof(float)));
        CUDA_CHECK(hipMallocManaged(&squares_sum, sizeof(float)));
        CUDA_CHECK(hipMallocManaged(&abs_normal_sum, sizeof(float)));

        *normal_sum = 0.0f;
        *squares_sum = 0.0f;
        *abs_normal_sum = 0.0f;

        // GPU kernel run
        int threads = 256;
        int blocks = (int)ceil(elems/256);
        meanStdkern<<<blocks, threads>>>(input, elems, mean, stddev, absMean, absStddev, normal_sum, squares_sum, abs_normal_sum);
        CUDA_CHECK(hipDeviceSynchronize()); // Synchronizes GPU and CPU memory

        // copy to the ret object
        ret.mean = *mean;
        ret.stddev = *stddev;
        ret.absMean = *absMean;
        ret.absStddev =  *absStddev;

        // Free the memory
        hipFree(mean);
        hipFree(stddev);
        hipFree(absMean);
        hipFree(absStddev);
        hipFree(normal_sum);
        hipFree(squares_sum);
        hipFree(abs_normal_sum);

        return ret;
    }

    __global__ void gpuPrinter(float * mem, size_t idx) {
        printf("Value at %d idx is %f\n", (int)idx, mem[idx]);
    }

    void gpuPrinterDispatch(float * mem, size_t idx) {
        gpuPrinter<<<1,1>>>(mem, idx);
    }

    __global__ void gpuPrinter(int32_t * mem, size_t idx) {
        printf("Value at %d idx is %d\n", (int)idx, (int)mem[idx]);
    }

    void gpuPrinterDispatch(int32_t * mem, size_t idx) {
        gpuPrinter<<<1,1>>>(mem, idx);
    }

    __global__ void gpuPrinter(int8_t * mem, size_t idx) {
        printf("Value at %d idx is %d\n", (int)idx, (int)mem[idx]);
    }

    void gpuPrinterDispatch(int8_t * mem, size_t idx) {
        gpuPrinter<<<1,1>>>(mem, idx);
    }

    void memCpyDevice(float * dest, float * source, size_t elems) {
        CUDA_CHECK(hipMemcpy(dest, source, elems*sizeof(float), hipMemcpyDeviceToDevice));
    }

    void memCpyDevice(int8_t * dest, int8_t * source, size_t elems) {
        CUDA_CHECK(hipMemcpy(dest, source, elems*sizeof(int8_t), hipMemcpyDeviceToDevice));
    }

    void memCpyHost(float * dest, float * source, size_t elems) {
        CUDA_CHECK(hipMemcpy(dest, source, elems*sizeof(float), hipMemcpyDeviceToHost));
    }

    void memCpyHost(int8_t * dest, int8_t * source, size_t elems) {
        CUDA_CHECK(hipMemcpy(dest, source, elems*sizeof(int8_t), hipMemcpyDeviceToHost));
    }

    void fieldSetGPU(float * gpuMem, float value) {
        float src = value;
        CUDA_CHECK(hipMemcpy(gpuMem, &src, 1*sizeof(float), hipMemcpyHostToDevice));
    }
/*
    float * unmanagedGPUAlloc(size_t num) {
        void * tmp;
        CUDA_CHECK(hipMalloc(&tmp, num*sizeof(float)));
        return (float *)tmp;
    }

    void unmanagedFree(float * in) {
        hipFree(in);
    }
*/
} // namespace integer
} // namespace gpu
} // namespace marian
