#include "hip/hip_runtime.h"
#include "cutlass/gemm/device/gemm.h"
#include "tensors/gpu/uint8tools.h"
#include "tensors/gpu/cuda_helpers.h"
#include <cmath>
//#include "/home/dheart/uni_stuff/postdoc/cutlass/include/cutlass/gemm/device/gemm.h"

namespace marian {

namespace hacky8bit {

inline std::string cutlassGetErrorString(cutlass::Status& status) {
    switch (status) {
        case cutlass::Status::kSuccess:
        return "Operation was successful.";
        case cutlass::Status::kErrorMisalignedOperand:
        return "Operands fail alignment requirements.";
        case cutlass::Status::kErrorInvalidLayout:
        return "Layout fails alignment requirement.";
        case cutlass::Status::kErrorInvalidProblem:
        return "Specified problem size is not supported by operator.";
        case cutlass::Status::kErrorNotSupported:
        return "Operation is not supported on current device.";
        case cutlass::Status::kErrorWorkspaceNull:
        return "The given workspace is null when it is required to be non-null";
        case cutlass::Status::kErrorInternal:
        return "An error within CUTLASS occurred.";
        case cutlass::Status::kInvalid:
        return "Status is unspecified.";
    }
    return "Unknown CUTLASS status. Update this section of the code.";
}

#define CUTLASS_CHECK(expr) do {                                             \
    cutlass::Status rc = (expr);                                        \
    ABORT_IF(rc != cutlass::Status::kSuccess,                                  \
                "Cutlass Error: {} - {}:{}: {}", cutlassGetErrorString(rc), __FILE__, __LINE__, #expr);  \
    } while(0)

    /*Cutlass matrices*/
    using ColumnMajor = cutlass::layout::ColumnMajor;
    using ColumnMajorT = cutlass::layout::RowMajor; //Transposing in cutlass is done by changing the input from RowMajor to ColumnMajor. Care of the output
    //using RowMajor = cutlass::layout::RowMajor;
    using CutlassGemmTT = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                    ColumnMajorT,  // Layout of A matrix
                                                    int8_t,        // Data-type of B matrix
                                                    ColumnMajorT,  // Layout of B matrix
                                                    int32_t,        // Data-type of C matrix
                                                    ColumnMajor>; // Layout of C matrix

    using CutlassGemmNT = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                    ColumnMajor,  // Layout of A matrix
                                                    int8_t,        // Data-type of B matrix
                                                    ColumnMajorT,  // Layout of B matrix
                                                    int32_t,        // Data-type of C matrix
                                                    ColumnMajor>; // Layout of C matrix

    using CutlassGemmTN = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                    ColumnMajorT,  // Layout of A matrix
                                                    int8_t,        // Data-type of B matrix
                                                    ColumnMajor,  // Layout of B matrix
                                                    int32_t,        // Data-type of C matrix
                                                    ColumnMajor>; // Layout of C matrix

    using CutlassGemmNN = cutlass::gemm::device::Gemm<int8_t,        // Data-type of A matrix
                                                    ColumnMajor,  // Layout of A matrix
                                                    int8_t,        // Data-type of B matrix
                                                    ColumnMajor,  // Layout of B matrix
                                                    int32_t,        // Data-type of C matrix
                                                    ColumnMajor>; // Layout of C matrix


    cutlass::Status cutlass_igemm_nn(bool transA, bool transB,
                     int M,
                     int N,
                     int K,
                     float alpha,
                     int8_t const *A,
                     int lda,
                     int8_t const *B,
                     int ldb,
                     float beta,
                     int32_t *C,
                     int ldc) {

        if (!transA && !transB) {
            CutlassGemmNN gemm_operator;
            CutlassGemmNN::Arguments args({M , N, K},  // Gemm Problem dimensions
                {A, lda},    // Tensor-ref for source matrix A
                {B, ldb},    // Tensor-ref for source matrix B
                {C, ldc},    // Tensor-ref for source matrix C
                {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                {alpha, beta}); // Scalars used in the Epilogue
            return gemm_operator(args);
        } else if (transA && !transB) {
            CutlassGemmTN gemm_operator;
            CutlassGemmTN::Arguments args({M , N, K},  // Gemm Problem dimensions
                {A, lda},    // Tensor-ref for source matrix A
                {B, ldb},    // Tensor-ref for source matrix B
                {C, ldc},    // Tensor-ref for source matrix C
                {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                {alpha, beta}); // Scalars used in the Epilogue
            return gemm_operator(args);
        } else if (!transA && transB) {
            CutlassGemmNT gemm_operator;
            CutlassGemmNT::Arguments args({M , N, K},  // Gemm Problem dimensions
                {A, lda},    // Tensor-ref for source matrix A
                {B, ldb},    // Tensor-ref for source matrix B
                {C, ldc},    // Tensor-ref for source matrix C
                {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                {alpha, beta}); // Scalars used in the Epilogue
            return gemm_operator(args);
        } else { // Final case (transA && transB)
            CutlassGemmTT gemm_operator;
            CutlassGemmTT::Arguments args({M , N, K},  // Gemm Problem dimensions
                {A, lda},    // Tensor-ref for source matrix A
                {B, ldb},    // Tensor-ref for source matrix B
                {C, ldc},    // Tensor-ref for source matrix C
                {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                {alpha, beta}); // Scalars used in the Epilogue
            return gemm_operator(args);
        }
    }


    template<class T>
    void sanityCheck(T * gpumem, size_t num_items, char typechar) {
        T * cpumem = new T[num_items];
        CUDA_CHECK(hipMemcpy(cpumem, gpumem, num_items*sizeof(T), hipMemcpyDeviceToHost));
        for (int i = 0; i < num_items; i++) {
            if (cpumem[i] == 127) {
                //fprintf(stderr, "Error at %d, type %c\n", i, typechar);
            }
        }
    }

    static inline int cols(const Tensor& tensor) { return tensor->shape()[-1]; }
    static inline int rows(const Tensor& tensor) { return tensor->shape().elements() / cols(tensor); }
/*
    static void unsetTensorMode(hipblasHandle_t cublasHandle) {
        cublasHandle; // fool warnings
      #if CUDA_VERSION >= 9000
        CUBLAS_CHECK(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));
      #endif
      }
        
    static void setTensorMode(hipblasHandle_t cublasHandle) {
        cublasHandle; // fool warnings
    #if CUDA_VERSION >= 9000
        static int mode = 0;  // 1: use TC; -1: do not use TC; 0: not set yet
        if (mode == 0) { // multi-thread note: this is sort-of thread-safe, since multiple threads would determine the same value
        const char* var = getenv("ENABLE_CUBLAS_TENSOR_OP_MATH_FP32");
        if (!var)
            var = "1";
        switch(var[0]) {
            case '0': mode = -1; break;
            case '1': mode =  1; break;
            default: ABORT("Invalid ENABLE_CUBLAS_TENSOR_OP_MATH_FP32={}", var);
        }
        if (mode > 0) { // try whether it can be set   --@TODO: check whether this actually works
            CUBLAS_CHECK(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
            hipblasMath_t actual = HIPBLAS_DEFAULT_MATH;
            hipblasGetMathMode(cublasHandle, &actual);
            if (actual != HIPBLAS_TENSOR_OP_MATH) {
            LOG(warn, "[gpu] TensorCores requested but not available");
            mode = -1;
            }
        }
        if (mode > 0)
            LOG(info, "[gpu] 16-bit TensorCores enabled for float32 matrix operations");
        }
        CUBLAS_CHECK(hipblasSetMathMode(cublasHandle, mode > 0 ? HIPBLAS_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH));
    #endif
    }*/

    __global__ void quantize(const float * input, int8_t * output, size_t items, float quantMult) {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        if (x < items) {
            output[x] = (int8_t)llrintf((input[x]*quantMult));
            //printf("%d Input: %f, output %d\n", x, input[x], (int)output[x]);
        }
    }
    
    template<class intType>
    __global__ void dequantize(intType * input, float * output, size_t items, float dequantMult) {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        if (x < items)
            output[x] = ((float)input[x])*dequantMult;
        //if (x == 0 || x == 1 || x ==3 ) {
        //    printf("Id %d  actual %f  mine: %f\n", x, output[x], ((float)input[x])*dequantMult);
        //    output[x] =  ((float)input[x])*dequantMult;
        //}
    }
    
    __global__ void findMaxMin(const float * input_gpu, int idxMax, int idxMin, float * output) {
        float absMax = abs(input_gpu[idxMax]);
        float absMin = abs(input_gpu[idxMin]);
        if (absMax > absMin) {
            output[0] = absMax;
        } else {
            output[0] = absMin;
        }
    }
    
    //@TODO rewrite with a nice singlePass GPU version that uses shared memory
    float maxAbs(hipblasHandle_t& handle, const float * input_gpu, size_t items, float * scratchMem) {
        //Get Max Absolute:
        int resMaxIdx;
        CUBLAS_CHECK(hipblasIsamax(handle, items, input_gpu, 1, &resMaxIdx));
        int resMinIdx;
        CUBLAS_CHECK(hipblasIsamin(handle, items, input_gpu, 1, &resMinIdx));
        float * output_gpu;
        if (scratchMem) {
            output_gpu = scratchMem;
        } else {
            CUDA_CHECK(hipMalloc(&output_gpu, 1*sizeof(float)));
        }
        findMaxMin<<<1,1>>>(input_gpu, resMaxIdx - 1, resMinIdx - 1, output_gpu); //FUCK YOU FORTRAN INDEXING
        float output;
        CUDA_CHECK(hipMemcpy(&output, &output_gpu[0], 1*sizeof(float), hipMemcpyDeviceToHost));
        if (!scratchMem) {
            CUDA_CHECK(hipFree(output_gpu));
        }
        return output;
    }

    __global__ void findMaxMin( float * input_gpu, int idxMax, int idxMin, float * output) {
        float absMax = abs(input_gpu[idxMax]);
        float absMin = abs(input_gpu[idxMin]);
        if (absMax > absMin) {
            output[0] = absMax;
        } else {
            output[0] = absMin;
        }
    }
    
    //@TODO rewrite with a nice singlePass GPU version that uses shared memory
    float maxAbs(hipblasHandle_t& handle, float * input_gpu, size_t items, float * scratchMem) {
        //Get Max Absolute:
        int resMaxIdx;
        CUBLAS_CHECK(hipblasIsamax(handle, items, input_gpu, 1, &resMaxIdx));
        int resMinIdx;
        CUBLAS_CHECK(hipblasIsamin(handle, items, input_gpu, 1, &resMinIdx));
        float * output_gpu;
        if (scratchMem) {
            output_gpu = scratchMem;
        } else {
            CUDA_CHECK(hipMalloc(&output_gpu, 1*sizeof(float)));
        }
        findMaxMin<<<1,1>>>(input_gpu, resMaxIdx - 1, resMinIdx - 1, output_gpu); //FUCK YOU FORTRAN INDEXING
        float output;
        CUDA_CHECK(hipMemcpy(&output, &output_gpu[0], 1*sizeof(float), hipMemcpyDeviceToHost));
        if (!scratchMem) {
            CUDA_CHECK(hipFree(output_gpu));
        }
        return output;
    }

    hipblasStatus_t cublas8bitGemmm(marian::Tensor& C,
        const marian::Tensor& A,
        const marian::Tensor& B,
        bool transA,
        bool transB,
        float beta,
        float scalar) {
        
        CUDA_CHECK(hipSetDevice((int)C->getDeviceId().no));
        auto backend = std::static_pointer_cast<gpu::Backend>(C->getBackend());
        //uint8_t * scratch = nullptr; //backend->getScratchMem();

        auto cublasHandle = backend->getCublasHandle();
    
        //First, We need to convert our tensors to 8bit, and use our scratch memory, so we don't do 10000 cudaMallocs.
        //But first, get it to work the stupid way:
        int32_t alpha_int = static_cast<int32_t>(scalar);
        int32_t beta_int = static_cast<int32_t>(beta);
       // fprintf(stderr, "Inside: Alpha is %d, beta is: %d\n", alpha_int, beta_int);
        int8_t* in8bitIntA;
        int8_t* in8bitIntB;
        int32_t * out32bitInt;

        CUDA_CHECK(hipMalloc(&out32bitInt, C->shape().elements()*sizeof(int32_t)));
        CUDA_CHECK(hipMalloc(&in8bitIntA, A->shape().elements()*sizeof(int8_t)));
        CUDA_CHECK(hipMalloc(&in8bitIntB, B->shape().elements()*sizeof(int8_t)));

        //CUDA_CHECK(hipMemset(in8bitIntB, 127, B->shape().elements()*sizeof(int8_t)));
        //CUDA_CHECK(hipMemset(in8bitIntA, 127, A->shape().elements()*sizeof(int8_t)));
        //CUDA_CHECK(hipMemset(out32bitInt, 127, C->shape().elements()*sizeof(int32_t)));
        //CUDA_CHECK(hipMemset(C->data<float>(), 127, C->shape().elements()*sizeof(float)));


        //Quantize:
        float aMaxAbs = maxAbs(cublasHandle, A->data<float>(), A->shape().elements(), nullptr/*reinterpret_cast<float *>(&scratch[0])*/);
        float bMaxAbs = maxAbs(cublasHandle, B->data<float>(), B->shape().elements(), nullptr/*reinterpret_cast<float *>(&scratch[4])*/);
        CUDA_CHECK(hipDeviceSynchronize());
        int rowsA = rows(A);
        int colsA = cols(A);
        int rowsB = rows(B);
        int colsB = cols(B);
        int rowsC = rows(C);
        int colsC = cols(C);
        if (colsA > 512) {
            std::swap(rowsA, colsA);
            if (colsA > 512) {
                fprintf(stderr, "Incompatible sizes: rows %d, cols %d\n", rowsA, colsA);
            }
        }

        if (colsB > 512) {
            std::swap(rowsB, colsB);
            if (colsB > 512) {
                fprintf(stderr, "Incompatible sizes: rows %d, cols %d\n", rowsB, colsB);
            }
        }

        if (colsC > 512) {
            std::swap(rowsC, colsC);
            if (colsC > 512) {
                fprintf(stderr, "Incompatible sizes: rows %d, cols %d\n", rowsC, colsC);
            }
        }
        quantize<<<rowsA, colsA>>>(A->data<float>(), in8bitIntA, A->shape().elements(), 127.0f/aMaxAbs);
        CUDA_CHECK(hipGetLastError());
        quantize<<<rowsB, colsB>>>(B->data<float>(), in8bitIntB, B->shape().elements(), 127.0f/bMaxAbs);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        sanityCheck(in8bitIntA, A->shape().elements(), 'q');
        sanityCheck(in8bitIntB, B->shape().elements(), 'w');
        /*
        for (int i = 0; i< A->shape().elements(); i++) {
            quantize<<<1, 1>>>(A->data<float>() + i, in8bitIntA, A->shape().elements(), 127.0f/aMaxAbs);
        }

        for (int i = 0; i< B->shape().elements(); i++) {
            quantize<<<1, 1>>>(B->data<float>() + i, in8bitIntB, B->shape().elements(), 127.0f/bMaxAbs);
        }*/


        //Things we do to setup lda, ldb, ldc and then multiply
      
        int m = A->shape().elements() / A->shape().back();
        int k = A->shape().back();
        if(transA)
          std::swap(m, k);
      
        int l = B->shape().elements() / B->shape().back();
        int n = B->shape().back();
        if(transB)
          std::swap(l, n);
      
        int lda = A->shape().back();
        int ldb = B->shape().back();
        int ldc = B->shape().back();
      
        if(transB)
          ldc = B->shape().elements() / B->shape().back();
        
        if (n * m != rows(C)*cols(C)) {
            fprintf(stderr, "n %d, m %d, crows %d, ccols %d\n", n, m, rows(C), cols(C));
        }
        hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        //fprintf(stderr, "m %d k %d l %d n %d lda %d ldb %d ldc %d transa %d transb %d, alpha_i %d, beta_i %d\n", m, k, l, n, lda, ldb, ldc, transA, transB, alpha_int, beta_int);
        //setTensorMode(cublasHandle);
        auto res = hipblasGemmEx(cublasHandle, /*1*/
            opB, /*2*/
            opA, /*3*/
            n, /*4*/
            m, /*5*/
            k, /*6*/
            &alpha_int, /*7*/
            in8bitIntB, /*8*/
            HIP_R_8I, /*9*/
            ldb, /*10*/
            in8bitIntA, /*11*/
            HIP_R_8I, /*12*/
            lda, /*13*/
            &beta_int, /*14*/
            out32bitInt, /*15*/
            HIP_R_32I, /*16*/
            ldc, /*17*/
            HIP_R_32I, /*18*/
            CUBLAS_GEMM_DEFAULT_TENSOR_OP); /*19*/
        CUBLAS_CHECK(res);
        CUDA_CHECK(hipDeviceSynchronize());
        //unsetTensorMode(cublasHandle);
        //fprintf(stderr, "after\n");
        //Convert back to float into C
        //fprintf(stderr, "C0 %d C1 %d C0 %d C1 %d\n", C->shape()[0], C->shape()[1], C->shape().elements() / C->shape().back(), C->shape().back());
        //fprintf(stderr, "rowsC %d, colsC %d\n", rowsC, colsC);
        if (C->shape().elements() != rowsC*colsC) {
            fprintf(stderr, "rowsC %d, colsC %d\n", rowsC, colsC);
        }
        dequantize<<<rowsC, colsC>>>(out32bitInt, C->data<float>(), C->shape().elements(), (aMaxAbs/127.0f)*(bMaxAbs/127.0f) );
        CUDA_CHECK(hipGetLastError());
        sanityCheck(C->data<float>(), C->shape().elements(), 'r');
        //for (int i = 0; i< C->shape().elements(); i++) {
        //    dequantize<<<1, 1>>>(out32bitInt +i, C->data<float>() +i, C->shape().elements(), (aMaxAbs/127.0f)*(bMaxAbs/127.0f) );
        //}
        CUDA_CHECK(hipDeviceSynchronize());
        //Free temporary used memory
        CUDA_CHECK(hipFree(out32bitInt));
        CUDA_CHECK(hipFree(in8bitIntA));
        CUDA_CHECK(hipFree(in8bitIntB));

        return res;
    }

    hipblasStatus_t cublas8bitGemmmEx(hipblasHandle_t handle,
        hipblasOperation_t transa, 
        hipblasOperation_t transb,
        int m, int n, int k,
        const float* alpha,
        const float* A, int lda,
        const float* B, int ldb,
        const float* beta,
        float* C, int ldc,
        bool useCutlass) {
            
        auto algorithm = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
        //if (m%4 == 0 && n % 4 == 0 && k % 4 ==0) {
        if (k % 4 ==0) {
            int BlockA = m;
            int ThreadsA = k;
            int BlockB = k;
            int ThreadsB = n;
            int BlockC = m;
            int ThreadsC = n;

            // Make sure that we have enough threads so that kernel launches don't fail
            if (ThreadsA > 512) {
                std::swap(BlockA, ThreadsA);
                if (ThreadsA > 512) {
                    //fprintf(stderr, "Incompatible sizes for A: rows %d, cols %d\n", BlockA, ThreadsA);
                    BlockA = (int)ceil((ThreadsA*BlockA)/512);
                    ThreadsA = 512;
                }
            }
    
            if (ThreadsB > 512) {
                std::swap(BlockB, ThreadsB);
                if (ThreadsB > 512) {
                    //fprintf(stderr, "Incompatible sizes for B: rows %d, cols %d\n", BlockB, ThreadsB);
                    BlockB = (int)ceil((ThreadsB*BlockB)/512);
                    ThreadsB = 512;
                }
            }
    
            if (ThreadsC > 512) {
                std::swap(BlockC, ThreadsC);
                if (ThreadsC > 512) {
                    //fprintf(stderr, "Incompatible sizes for C: rows %d, cols %d\n", BlockC, ThreadsC);
                    BlockC = (int)ceil((ThreadsC*BlockC)/512);
                    ThreadsC = 512;
                }
            }

            int32_t alpha_int = static_cast<int32_t>(*alpha);
            int32_t beta_int = static_cast<int32_t>(*beta);
            int8_t* in8bitIntA;
            int8_t* in8bitIntB;
            int32_t * out32bitInt;

            CUDA_CHECK(hipMalloc(&out32bitInt, m*n*sizeof(int32_t)));
            CUDA_CHECK(hipMalloc(&in8bitIntA, m*k*sizeof(int8_t)));
            CUDA_CHECK(hipMalloc(&in8bitIntB, k*n*sizeof(int8_t)));

            float aMaxAbs = maxAbs(handle, A, m*k, nullptr/*reinterpret_cast<float *>(&scratch[0])*/);
            float bMaxAbs = maxAbs(handle, B, k*n, nullptr/*reinterpret_cast<float *>(&scratch[4])*/);

            quantize<<<BlockA, ThreadsA>>>(A, in8bitIntA, m*k, 127.0f/aMaxAbs);
            quantize<<<BlockB, ThreadsB>>>(B, in8bitIntB, k*n, 127.0f/bMaxAbs);
            CUDA_CHECK(hipDeviceSynchronize());

            hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
            if (useCutlass) {
                bool transACutlass = false;
                bool transBCutlass = false;
                if (transa == HIPBLAS_OP_T)
                    transACutlass = true;

                if (transb == HIPBLAS_OP_T)
                    transBCutlass = true;

                CUTLASS_CHECK(cutlass_igemm_nn(transACutlass, transBCutlass,
                    m, n, k, *alpha,
                    in8bitIntA, lda,
                    in8bitIntB, ldb, *beta,
                    out32bitInt, ldc));
            } else {
                auto res = hipblasGemmEx(handle, transa, transb,
                    m, n, k, &alpha_int,
                    in8bitIntA, HIP_R_8I, lda,
                    in8bitIntB, HIP_R_8I, ldb, &beta_int,
                    out32bitInt, HIP_R_32I, ldc,
                    HIP_R_32I, algorithm);
            }
            
            dequantize<<<BlockC, ThreadsC>>>(out32bitInt, C, m*n, (aMaxAbs/127.0f)*(bMaxAbs/127.0f) );
            CUDA_CHECK(hipDeviceSynchronize());
            //Free temporary used memory
            CUDA_CHECK(hipFree(out32bitInt));
            CUDA_CHECK(hipFree(in8bitIntA));
            CUDA_CHECK(hipFree(in8bitIntB));
            return res;
        } else {
            auto res = hipblasGemmEx(handle, transa, transb,
                m, n, k, alpha,
                A, HIP_R_32F, lda,
                B, HIP_R_32F, ldb, beta,
                C, HIP_R_32F, ldc,
                HIP_R_32F, algorithm);
            return res;
        }

    }


} // namespace hacky8bit
} //namespace marian