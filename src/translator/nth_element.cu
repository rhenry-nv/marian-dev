#include "hip/hip_runtime.h"
/* All or part of this file was contributed by Intel under license:
 *   Copyright (C) 2017-2018 Intel Corporation
 *   SPDX-License-Identifier: MIT
 */

 /* All or part of this file was contributed by NVIDIA under license:
 *   Copyright (C) 2020 NVIDIA Corporation
 *   SPDX-License-Identifier: MIT
 */

#include <iostream>

#include "translator/nth_element.h"
#include "3rd_party/topk.cuh"

#include <hip/hip_runtime.h>
#include "tensors/gpu/cuda_helpers.h"

namespace marian {
class NthElementGPU {
public:
  NthElementGPU() = delete;
  NthElementGPU(const NthElementGPU& copy) = delete;

  NthElementGPU(size_t maxBeamSize,
                size_t maxBatchSize,
                DeviceId deviceId)
      : deviceId_(deviceId),
        maxBeamSize_(maxBeamSize), maxBatchSize_(maxBatchSize) {
    // std::cerr << "NthElement::NthElement" << std::endl;

    hipSetDevice(deviceId_.no);

    const int tempElts = maxBatchSize * maxBeamSize * maxBeamSize * MAX_BLOCKS_PER_BEAM;
    CUDA_CHECK(hipMalloc((void**)&topk_tmp_id_buf, tempElts * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&topk_tmp_val_buf, tempElts * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&tops, maxBatchSize * maxBeamSize * sizeof(TopK)));
    CUDA_CHECK(hipHostAlloc((void**)&topsHost, maxBeamSize * maxBatchSize * sizeof(TopK), hipHostMallocDefault));
  }

  ~NthElementGPU() {
    // No CUDA error checking as this is a destructor and we cannot do anything about errors anyway.
    hipSetDevice(deviceId_.no);
    hipFree(topk_tmp_id_buf);
    hipFree(topk_tmp_val_buf);
    hipFree(tops);
    hipHostFree(topsHost);
  }

private:
  template <typename T>
  void selectNBest(T* probs, 
                   const int batchSize,
                   const int beamsPerBatch,
                   const int beamWidth,
                   const int vocabSize) {
    hipSetDevice(deviceId_.no);

    topK_kernelLauncher(probs, topk_tmp_id_buf, topk_tmp_val_buf, tops,
                        batchSize, beamsPerBatch, beamWidth, vocabSize, 0);    
  }

public:
  void getNBestList(Tensor scores,
                    size_t N,
                    std::vector<float>& outCosts,
                    std::vector<unsigned>& outKeys,
                    const bool isFirst) {
    hipSetDevice(deviceId_.no);

    const auto vocabSize = scores->shape()[-1];
    const auto inputN    = scores->shape()[-2];
    const auto dimBatch  = scores->shape()[-4];

    ABORT_IF(inputN != (isFirst ? 1 : N), "Input tensor has wrong beam dim??"); // @TODO: Remove isFirst argument altogether
    ABORT_IF(vocabSize > MAX_VOCAB_SIZE, "GetNBestList(): actual vocab size {} exceeds MAX_VOCAB_SIZE of {}", vocabSize, MAX_VOCAB_SIZE);
    ABORT_IF(dimBatch > maxBatchSize_, "GetNBestList(): actual batch size {} exceeds initialization parameter {}", dimBatch, maxBatchSize_);
    ABORT_IF(std::max(N, (size_t)inputN) > maxBeamSize_, "GetNBestList(): actual beam size {} exceeds initialization parameter {}", N, maxBeamSize_);

    if(scores->type() == Type::float32) {
      selectNBest(scores->data<float>(), dimBatch, inputN, N, vocabSize);
#if COMPILE_FP16
    } else if(scores->type() == Type::float16) {
      selectNBest(scores->data<half>(), dimBatch, inputN, N, vocabSize);
#endif
    } else {
      ABORT("getNBestList not implemented for type {}", scores->type());
    }
    getPairs(dimBatch * N, outKeys, outCosts);
    ABORT_IF(outKeys.size() != dimBatch * N, "Expected {} but got {} values during topk", outKeys.size(), dimBatch * N);
  }

private:
  void getPairs(size_t numElts,
                std::vector<unsigned>& outKeys,
                std::vector<float>& outValues) {
    hipSetDevice(deviceId_.no);
    CUDA_CHECK(hipMemcpyAsync(topsHost,
                               tops,
                               numElts * sizeof(TopK),
                               hipMemcpyDeviceToHost,
                               /* stream_ */ 0));

    CUDA_CHECK(hipStreamSynchronize(/* stream_ */ 0));

    for(size_t i = 0; i < numElts; ++i) {
      outKeys.push_back(topsHost[i].p);
      outValues.push_back(topsHost[i].u);
    }
  }

  DeviceId deviceId_;

  const int MAX_VOCAB_SIZE = 500000;
  size_t maxBeamSize_;
  size_t maxBatchSize_;

  int* topk_tmp_id_buf; // [maxBatchSize * maxBeamSize, maxBeamSize * MAX_BLOCKS_PER_BEAM]
  float* topk_tmp_val_buf; // [maxBatchSize * maxBeamSize, maxBeamSize * MAX_BLOCKS_PER_BEAM]
  TopK* tops; // [maxBatchSize, maxBeamSize]
  TopK* topsHost; // [maxBatchSize, maxBeamSize]
};

// factory function
// Returns a lambda with the same signature as the getNBestList() function.
GetNBestListFn createGetNBestListGPUFn(size_t beamSize, size_t dimBatch, DeviceId deviceId) {
  auto nth = New<NthElementGPU>(beamSize, dimBatch, deviceId);
  return [nth](Tensor logProbs, size_t N, std::vector<float>& outCosts, std::vector<unsigned>& outKeys, const bool isFirst) {
    return nth->getNBestList(logProbs, N, outCosts, outKeys, isFirst);
  };
}

}  // namespace marian
